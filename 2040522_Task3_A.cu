#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <time.h>//Time heading
 
//Password Cracking using CUDA 
__device__ char* encryptDecrypt(char* tempPassword){

	char * generatedPwd = (char *) malloc(sizeof(char) * 11);

	generatedPwd[0] = tempPassword[0] + 2;
	generatedPwd[1] = tempPassword[0] - 2;
	generatedPwd[2] = tempPassword[0] + 1;
	generatedPwd[3] = tempPassword[1] + 3;
	generatedPwd[4] = tempPassword[1] - 3;
	generatedPwd[5] = tempPassword[1] - 1;
	generatedPwd[6] = tempPassword[2] + 2;
	generatedPwd[7] = tempPassword[2] - 2;
	generatedPwd[8] = tempPassword[3] + 4;
	generatedPwd[9] = tempPassword[3] - 4;
	generatedPwd[10] = '\0';

	for(int i =0; i<10; i++){
		if(i >= 0 && i < 6){ 
			if(generatedPwd[i] > 122){
				generatedPwd[i] = (generatedPwd[i] - 122) + 97;
			}else if(generatedPwd[i] < 97){
				generatedPwd[i] = (97 - generatedPwd[i]) + 97;
			}
		}else{ 
			if(generatedPwd[i] > 57){
				generatedPwd[i] = (generatedPwd[i] - 57) + 48;
			}else if(generatedPwd[i] < 48){
				generatedPwd[i] = (48 - generatedPwd[i]) + 48;
			}
		}
	}
	return generatedPwd;
}

__global__ void crack(char * alphabet, char * numbers){

char matchedPwd[4];

matchedPwd[0] = alphabet[blockIdx.x];
matchedPwd[1] = alphabet[blockIdx.y];

matchedPwd[2] = numbers[threadIdx.x];
matchedPwd[3] = numbers[threadIdx.y];


char* encryptedPwd = "plodwy3171"; //nz13
char* search = encryptDecrypt(matchedPwd);
int iter = 0;
int is_match = 0;
while (*encryptedPwd != '\0' || *search != '\0') {
	if (*encryptedPwd == *search) {
		encryptedPwd++;
		search++;
	} else if ((*encryptedPwd == '\0' && *search != '\0') || (*encryptedPwd != '\0' && *search == '\0') || *encryptedPwd != *search) {
		is_match = 1;
	
		break;
	}
}
if (is_match == 0) {
	printf("Password found successfully: %c%c%c%c \n", matchedPwd[0],matchedPwd[1],matchedPwd[2],matchedPwd[3]);
}


}

int time_count(struct timespec *start, struct timespec *end,
                   long long int *diff)
{
    long long int in_sec = end->tv_sec - start->tv_sec;
    long long int in_nano = end->tv_nsec - start->tv_nsec;
    if (in_nano < 0)
    {
        in_sec--;
        in_nano += 1000000000;
    }
    *diff = in_sec * 1000000000 + in_nano;
    return !(*diff > 0);
}

int main(int argc, char ** argv){

    struct timespec start, end;
    long long int time_used;

	char cpuCharacter[26] = {'a','b','c','d','e','f','g','h','i','j','k','l','m','n','o','p','q','r','s','t','u','v','w','x','y','z'};
	char cpuDigits[26] = {'0','1','2','3','4','5','6','7','8','9'};

	char * gpuCharacter;
	hipMalloc( (void**) &gpuCharacter, sizeof(char) * 26); 
	hipMemcpy(gpuCharacter, cpuCharacter, sizeof(char) * 26, hipMemcpyHostToDevice);

	char * gpuDigits;
	hipMalloc( (void**) &gpuDigits, sizeof(char) * 26); 
	hipMemcpy(gpuDigits, cpuDigits, sizeof(char) * 26, hipMemcpyHostToDevice);
    
    clock_gettime(CLOCK_MONOTONIC_RAW, &start);
	crack<<< dim3(26,26,1), dim3(10,10,1) >>>( gpuCharacter, gpuDigits );
    hipDeviceSynchronize();

    clock_gettime(CLOCK_MONOTONIC_RAW, &end);
    time_count(&start, &end, &time_used);

    printf("Time taken: %f seconds OR %lld Nano Seconds\n", (time_used / 1.0e9), (time_used));
  
    
	return 0;
}












